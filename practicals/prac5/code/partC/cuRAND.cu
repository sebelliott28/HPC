//random number generation

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

int main ( void ) {

//Allocate pointers for host and device memory
float *h_input, *h_output;
float *d_input, *d_output;

//malloc() host memory on local RAM
h_input = (float*) malloc(mem_size);
h_output = (float*) malloc(mem_size);

//allocate device memory input and output arrays
hipMalloc( (void**) &d_input, mem_size);
hipMalloc( (void**) &d_output, mem_size);

hipMemcpy(d_input, h_input, mem_size, hipMemcpyHostToDevice);

//Declare random variable
hiprandGenerator_t gen;

//Create random number generator 
hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

//Set generator options
hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

//Generator random numbers
hiprandGenerateNormal (gen, d_input, NUM_ELS, 0.0f, 1.0f); 

hipMemcpy(h_output, d_output, mem_size, hipMemcpyDeviceToHost);

//Cleanup memory

free(h_input);
free(h_output);
hipFree(d_input);
hipFree(d_output);

}

