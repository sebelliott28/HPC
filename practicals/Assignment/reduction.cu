
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <sys/time.h>

#define MILLION 1000000.0

#define NUM_ELS 1024

__global__ void reduction(int num_els, float *d_input, float *d_output)
{
    // Allocate shared memory

    __shared__  float smem_array[NUM_ELS];

    //reset d_output to 0

    d_output[0] = 0;

    //define local thread id for each block, and input index across all blocks

    int tid = threadIdx.x;
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    // first, each thread loads data into shared memory

    if (index < num_els) {
    	smem_array[tid] = d_input[index];
    }
    else smem_array[tid] =0;

    // next, we perform binary tree reduction

    for (int d = blockDim.x/2; d > 0; d /= 2) {
      __syncthreads();  // ensure previous step completed 
      if (tid<d)  smem_array[tid] += smem_array[tid+d];
    }

    // finally, first thread in each block puts result into global memory
    
    if (tid == 0) atomicAdd(&d_output[0], smem_array[0]);
}


// wall_clock_time - wall clock time function 

double wall_clock_time (void) {

  double secs;
  struct timeval tp;

  gettimeofday (&tp,NULL);
  secs = (MILLION * (double) tp.tv_sec + (double) tp.tv_usec) / MILLION;
  return secs;

}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int main( int argc, const char** argv) 
{
    int num_els, num_threads, mem_size, nblocks;

    float *h_data;
    float *d_input, *d_output;
 
    double start, finish;

   //Prompt user to enter number of numbers to work with
    printf("Enter the number of random numbers to work with:\t");
    scanf("%d", &num_els);

    // initialise card

    num_threads = NUM_ELS;
    mem_size    = sizeof(float) * num_els;


    // allocate device memory input and output arrays, and host memory

    hipMalloc((void**)&d_input, mem_size);
    hipMalloc((void**)&d_output, sizeof(float));
    h_data = (float*) malloc(mem_size);

    // now need to generate random numbers into d_input using cuRAND library
    
    // initiate random number generator
    hiprandGenerator_t gen;

    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

    // Set the generator options
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    // create random numbers, initialized to h_data, with mean 500 & standard deviation 200 
    hiprandGenerateNormal(gen, d_input, num_els, 500.0f, 50.0f);

    // execute the kernel

    nblocks = num_els/NUM_ELS + (num_els % NUM_ELS == 0 ? 0 : 1); //effective ceiling function to calculate number of blocks required
    printf("nblocks = %d\n", nblocks); //check here

    start = wall_clock_time (); 

    reduction<<<nblocks,num_threads>>>(num_els, d_input,d_output);

    finish = wall_clock_time ();

    // copy result from device to host

    hipMemcpy(h_data, d_output, sizeof(float), hipMemcpyDeviceToHost);

    // check results

    printf("reduction error = %f\n",h_data[0]/num_els);
    printf("process time = %e s\n", finish - start);

    // cleanup memory

    free(h_data);
    hipFree(d_input);
    hipFree(d_output);

    // CUDA exit -- needed to flush printf write buffer

    hipDeviceReset();
}
