// In this assignment you will write a basic kernel where every thread 
// will write out to console string "Hello world!".
// You will also initialize GPU using cudaSetDevice() and also launch
// your "Hello world" kernel.

#include <stdio.h>
#include <stdlib.h>
// we have to include few more things
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

//----------------------------------------------------------------------
// TASK 2: Write a "Hello world" kernel
// 
// Remember that kernel is defined by __global__ and inside it looks like
// a serial code for CPU. For printing out to console you can use printf().

// write your kernel here

__global__ void helloWorld_GPU(void) {
	printf("hello world!\n");
}

//----------------------------------------------------------------------

int main(void) {
  //----------------------------------------------------------------------
  // TASK 1: Initiate GPU using cudaSetDevice()
  //
  // You can also try to write a check if there is a device with that id,
  // so the code behaves nicely when it fails
  
  // write you GPU initialization here

	int deviceid = 0;
	int devCount;

	hipGetDeviceCount(&devCount);

	if (deviceid<devCount) {
		hipSetDevice(deviceid);
	}
	else return(1);


  //----------------------------------------------------------------------

  
  
  //----------------------------------------------------------------------
  // TASK 3: execute your "Hello world" kernel on 1 block with 5 threads 
  //         using execution configuration syntax.
  // 
  // You may use whatever syntax version you prefer, a simplified one 
  // dimensional or full three dimensional call using dim3 data type.
  
  // execute your "Hello world" kernel here

int nBlocks = 1;
int nThreads = 5;

helloWorld_GPU<<< nBlocks, nThreads >>>();
  
  //----------------------------------------------------------------------
 
  hipDeviceReset(); 
  return (0);
}
