// In this assignment you will write a kernel for decimation of an integer 
// array by factor of two (which sums two neighbouring integer numbers into
// one). More description is in notes for this practical.

// Your tasks are:
// 1) to calculate decimation by factor of two using GPU and store 
//    the result to the host.
// 2) to calculate decimation by factor of four using GPU and store 
//    the result to the host.
// To do that you have to:
// 1) write the host code to manage memory, data initialization and data 
//    transfers
// 2) write GPU kernel which is appropriate for the tasks
// 3) configure grid for your GPU kernel

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <math.h>

#define M 32

int Compare_results(int *GPU_output, int *h_input, size_t size, int DIT_factor){
	int itemp;
	size_t half_size;
	int error = 0;
	int nErrors = 0;
	int cislo = 0;
	
	half_size = size/DIT_factor;
	for(size_t f=0; f<half_size; f++){
		itemp=0;
		for(int t=0; t<DIT_factor; t++){
			itemp = itemp + h_input[f*DIT_factor + t];
		}
		error = (GPU_output[f]-itemp);
		if(error!=0 && cislo<20) {
			printf("f: %d; error: %d; GPU: %d; CPU: %d\n", f, error, GPU_output[f], itemp);
			cislo++;
		}
		if(error!=0) nErrors++;
	}
	
	return(nErrors);
}

void Check_errors(int *GPU_sum2, int *GPU_sum4, int *h_input, size_t size){
	int nErrors = 0;
	printf("Checking result for sum of two elements...\n");
	nErrors = Compare_results(GPU_sum2, h_input, size, 2);
	if(nErrors>0) printf("Correct solution and your solution does not match!\n");
	else printf("Your solution is correct!\n");
	
	printf("Checking result for sum of four elements...\n");
	nErrors = Compare_results(GPU_sum4, h_input, size, 4);
	if(nErrors>0) printf("Correct solution and your solution does not match!\n");
	else printf("Your solution is correct!\n");	
}

//----------------------------------------------------------------------
// TASK: Write your decimation GPU kernel
// When writing your kernel think carefully what each thread needs to read
// and what is it writes out and to where.
// Also consider what other threads might be reading and writing and where.

// put your kernel here

__global__ void decimation(int *d_GPU_sum2, int *d_GPU_sum4, int *d_input) {
	
	for (int f=0; f<M; f+=2) {
		int index2=blockIdx.x*blockDim.x + threadIdx.x;
		int index_out2=floor(f/2);
		d_GPU_sum2[index_out2] = d_input[index2] + d_input[index2+1]; 
	}

	for (int g=0; g<M; g+=4) {
		int index4=blockIdx.x*blockDim.x + threadIdx.x;
		int index_out4=floor(g/4);
		d_GPU_sum4[index_out4] = d_input[index4] + d_input[index4+1] + d_input[index4+2] + d_input[index4+3];
	}
		
}

//----------------------------------------------------------------------



int main(void) {
	// Size of the array
	size_t N = 67108864;
	// Declaration of arrays used in this practical
	int *h_GPU_sum2, *h_GPU_sum4, *h_input;
	
	// memory allocation
	h_GPU_sum2 = (int*) malloc(N*sizeof(*h_GPU_sum2));
	h_GPU_sum4 = (int*) malloc(N*sizeof(*h_GPU_sum4));
	h_input       = (int*) malloc(N*sizeof(*h_input));
	if(h_GPU_sum2==NULL || h_GPU_sum4==NULL || h_input==NULL) return(1);
	
	// initiate host data
	for(size_t f=0; f<N; f++) {
		h_input[f] = f;
	}
	
	//----------------------------------------------------------------------
	// TASK: Write host code
	// Do not forget to initialize the device.
	// Remember that configuration of your grid depends on how you have 
	// written your decimation kernel
	
	// finish the host code here
	

int deviceid = 0;
int devCount;
hipGetDeviceCount(&devCount);
if(deviceid<devCount) hipSetDevice(deviceid);
else return(1);

	int *d_GPU_sum2, *d_GPU_sum4, *d_input;
	
	hipMalloc(&d_GPU_sum2, N*sizeof(*d_GPU_sum2));
	hipMalloc(&d_GPU_sum4,	N*sizeof(*d_GPU_sum4));
	hipMalloc(&d_input, N*sizeof(*d_input));

hipMemcpy(d_input, h_input, N*sizeof(*h_input), hipMemcpyHostToDevice);


decimation<<<N/(M*M),M>>>(d_GPU_sum2, d_GPU_sum4, d_input);
	
hipMemcpy(h_GPU_sum2, d_GPU_sum2,  N*sizeof(*d_GPU_sum2), hipMemcpyDeviceToHost); 
hipMemcpy(h_GPU_sum4, d_GPU_sum4,  N*sizeof(*d_GPU_sum4), hipMemcpyDeviceToHost);
	//----------------------------------------------------------------------
	
	Check_errors(h_GPU_sum2, h_GPU_sum4, h_input, N);
	
	// free memory on the host and the device

	hipFree(d_GPU_sum2);
	hipFree(d_GPU_sum4);
	hipFree(d_input);
	free(h_GPU_sum2);
	free(h_GPU_sum4);
	free(h_input);
	
	hipDeviceReset();
	return (0);
}
